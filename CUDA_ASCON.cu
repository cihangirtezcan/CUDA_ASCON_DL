#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <Windows.h>
#include "rdrand.h"
#include <intrin.h>
#include <immintrin.h>
#include <string.h>
#include <ctime>

#define BLOCKS				256
#define THREADS				1024
#define TRIALS				1024*4
//__int64 trial = 1024*4, keys = 100;
__int64 trial = 1, keys = 100;
int rotation = 0;
double PCFreq = 0.0;
__int64 CounterStart = 0;

#define RDRAND_MASK	0x40000000
#define RETRY_LIMIT 10
#ifdef _WIN64
typedef uint64_t _wordlen_t;
#else
typedef uint32_t _wordlen_t;
#endif
#define bit32 unsigned int
#define bit64 unsigned __int64 
bit64 key[2], *key_d, *nonce, *nonce_d, *IV_d, *keyrows_d, *keyrows, *IV;
bit64 state[5] = { 0 }, t[5] = { 0 };
bit64 constants[16] = { 0xf0, 0xe1, 0xd2, 0xc3, 0xb4, 0xa5, 0x96, 0x87, 0x78, 0x69, 0x5a, 0x4b, 0x3c, 0x2d, 0x1e, 0x0f };
int key_choice = 0;
void StartCounter(){
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		printf("QueryPerformanceFrequency failed!\n");

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter(){
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}
void print_state(bit64 state[5]) {	for (int i = 0; i < 5; i++) printf("%016I64x\n", state[i]);}
void add_constant(bit64 state[5], int i, int a) {	state[2] = state[2] ^ constants[12 - a + i];}
void sbox(bit64 x[5]) {
	x[0] ^= x[4]; x[4] ^= x[3]; x[2] ^= x[1];
	t[0] = x[0]; t[1] = x[1]; t[2] = x[2]; t[3] = x[3]; t[4] = x[4];
	t[0] = ~t[0]; t[1] = ~t[1]; t[2] = ~t[2]; t[3] = ~t[3]; t[4] = ~t[4];
	t[0] &= x[1]; t[1] &= x[2]; t[2] &= x[3]; t[3] &= x[4]; t[4] &= x[0];
	x[0] ^= t[1]; x[1] ^= t[2]; x[2] ^= t[3]; x[3] ^= t[4]; x[4] ^= t[0];
	x[1] ^= x[0]; x[0] ^= x[4]; x[3] ^= x[2]; x[2] = ~x[2];
}
bit64 rotate(bit64 x, int l) {
	bit64 temp;
	temp = (x >> l) ^ (x << (64 - l));
	return temp;
}
__device__ bit64 rotater(bit64 x, int l) {
	bit64 temp;
	temp = (x >> l) ^ (x << (64 - l));
	return temp;
}
void linear(bit64 state[5]) {
	bit64 temp0, temp1;
	temp0 = rotate(state[0], 19);
	temp1 = rotate(state[0], 28);
	state[0] ^= temp0 ^ temp1;
	temp0 = rotate(state[1], 61);
	temp1 = rotate(state[1], 39);
	state[1] ^= temp0 ^ temp1;
	temp0 = rotate(state[2], 1);
	temp1 = rotate(state[2], 6);
	state[2] ^= temp0 ^ temp1;
	temp0 = rotate(state[3], 10);
	temp1 = rotate(state[3], 17);
	state[3] ^= temp0 ^ temp1;
	temp0 = rotate(state[4], 7);
	temp1 = rotate(state[4], 41);
	state[4] ^= temp0 ^ temp1;
}
void p(bit64 state[5], int a) {
	for (int i = 0; i < a; i++) {
		add_constant(state, i, a);
		sbox(state);
		linear(state);
	}
}
void initialization(bit64 state[5], bit64 key[2]) {
	p(state, 12);
	state[3] ^= key[0];
	state[4] ^= key[1];
}
void encrypt(bit64 state[5], int length, bit64 plaintext[], bit64 ciphertext[]) {
	ciphertext[0] = plaintext[0] ^ state[0];
	for (int i = 1; i < length; i++) {
		p(state, 6);
		ciphertext[i] = plaintext[i] ^ state[0];
		state[0] = plaintext[i] ^ state[0];
	}
}
void decrypt(bit64 state[5], int length, bit64 plaintext[], bit64 ciphertext[]) {
	ciphertext[0] = plaintext[0] ^ state[0];
	for (int i = 1; i < length; i++) {
		p(state, 6);
		ciphertext[i] = plaintext[i] ^ state[0];
		state[0] = plaintext[i];
	}
}
void main_old() {
	bit64 IV = 0x80400c0600000000, key[2] = { 0xffffffffffffffff, 0x123456789abcdef0 }, nonce[2] = { 0 };
	bit64 plaintext[10] = { 0 }, ciphertext[10];
	state[0] = IV;
	state[1] = key[0];
	state[2] = key[1];
	state[3] = nonce[0];
	state[4] = nonce[1];
	print_state(state); printf("\n");
	//p(state, 12);
	initialization(state, key);
	print_state(state);
	encrypt(state, 10, plaintext, ciphertext); printf("\n");
	for (int i = 0; i < 10; i++) printf("%016I64x\n", ciphertext[i]);
	state[0] = IV;
	state[1] = key[0];
	state[2] = key[1];
	state[3] = nonce[0];
	state[4] = nonce[1]; printf("\n");
	print_state(state); printf("\n");
	//p(state, 12);
	initialization(state, key);
	print_state(state);
	decrypt(state, 10, ciphertext, plaintext); printf("\n");
	for (int i = 0; i < 10; i++) printf("%016I64x\n", plaintext[i]);
}
/*! \brief Queries cpuid to see if rdrand is supported
*
* rdrand support in a CPU is determined by examining the 30th bit of the ecx
* register after calling cpuid.
*
* \return bool of whether or not rdrand is supported
*/
int RdRand_cpuid() {
	int info[4] = { -1, -1, -1, -1 };
	/* Are we on an Intel processor? */
	__cpuid(info, /*feature bits*/0);
	if (memcmp((void *)&info[1], (void *) "Genu", 4) != 0 ||
		memcmp((void *)&info[3], (void *) "ineI", 4) != 0 ||
		memcmp((void *)&info[2], (void *) "ntel", 4) != 0) {
		return 0;
	}
	/* Do we have RDRAND? */
	__cpuid(info, /*feature bits*/1);
	int ecx = info[2];
	if ((ecx & RDRAND_MASK) == RDRAND_MASK)
		return 1;
	else
		return 0;
}
/*! \brief Determines whether or not rdrand is supported by the CPU
*
* This function simply serves as a cache of the result provided by cpuid,
* since calling cpuid is so expensive. The result is stored in a static
* variable to save from calling cpuid on each invocation of rdrand.
*
* \return bool/int of whether or not rdrand is supported
*/
int RdRand_isSupported() {
	static int supported = RDRAND_SUPPORT_UNKNOWN;

	if (supported == RDRAND_SUPPORT_UNKNOWN)
	{
		if (RdRand_cpuid())
			supported = RDRAND_SUPPORTED;
		else
			supported = RDRAND_UNSUPPORTED;
	}

	return (supported == RDRAND_SUPPORTED) ? 1 : 0;
}
int rdrand_16(uint16_t* x, int retry) {
	if (RdRand_isSupported())	{
		if (retry)		{
			for (int i = 0; i < RETRY_LIMIT; i++)			{
				if (_rdrand16_step(x))
					return RDRAND_SUCCESS;
			}
			return RDRAND_NOT_READY;
		}
		else		{
			if (_rdrand16_step(x))				return RDRAND_SUCCESS;
			else				return RDRAND_NOT_READY;
		}
	}
	else	{		return RDRAND_UNSUPPORTED;	}
}
int rdrand_32(uint32_t* x, int retry) {
	if (RdRand_isSupported())	{
		if (retry)		{
			for (int i = 0; i < RETRY_LIMIT; i++)			{
				if (_rdrand32_step(x))
					return RDRAND_SUCCESS;
			}
			return RDRAND_NOT_READY;
		}
		else		{
			if (_rdrand32_step(x))
				return RDRAND_SUCCESS;
			else
				return RDRAND_NOT_READY;
		}
	}
	else	{		return RDRAND_UNSUPPORTED;	}
}
int rdrand_64(uint64_t* x, int retry) {
	if (RdRand_isSupported())	{
		if (retry)		{
			for (int i = 0; i < RETRY_LIMIT; i++)			{
				if (_rdrand64_step(x))
					return RDRAND_SUCCESS;
			}
			return RDRAND_NOT_READY;
		}
		else		{
			if (_rdrand64_step(x))				return RDRAND_SUCCESS;
			else				return RDRAND_NOT_READY;
		}
	}
	else	{		return RDRAND_UNSUPPORTED;	}
}
int rdrand_get_n_64(unsigned int n, uint64_t *dest) {
	int success;
	int count;
	unsigned int i;

	for (i = 0; i<n; i++) 	{
		count = 0;
		do 		{
			success = rdrand_64(dest, 1);
		} while ((success == 0) && (count++ < RETRY_LIMIT));
		if (success != RDRAND_SUCCESS) return success;
		dest = &(dest[1]);
	}
	return RDRAND_SUCCESS;
}
int rdrand_get_n_32(unsigned int n, uint32_t *dest) {
	int success;
	int count;
	unsigned int i;
	for (i = 0; i<n; i++) 	{
		count = 0;
		do 		{
			success = rdrand_32(dest, 1);
		} while ((success == 0) && (count++ < RETRY_LIMIT));
		if (success != RDRAND_SUCCESS) return success;
		dest = &(dest[1]);
	}
	return RDRAND_SUCCESS;
}
int rdrand_get_bytes(unsigned int n, unsigned char *dest) {
	unsigned char *start;
	unsigned char *residualstart;
	_wordlen_t *blockstart;
	_wordlen_t i, temprand;
	unsigned int count;
	unsigned int residual;
	unsigned int startlen;
	unsigned int length;
	int success;

	/* Compute the address of the first 32- or 64- bit aligned block in the destination buffer, depending on whether we are in 32- or 64-bit mode */
	start = dest;
	if (((uint32_t)start % (uint32_t) sizeof(_wordlen_t)) == 0) 	{
		blockstart = (_wordlen_t *)start;
		count = n;
		startlen = 0;
	}
	else 	{
		blockstart = (_wordlen_t *)(((_wordlen_t)start & ~(_wordlen_t)(sizeof(_wordlen_t)-1)) + (_wordlen_t)sizeof(_wordlen_t));
		count = n - (sizeof(_wordlen_t)-(unsigned int)((_wordlen_t)start % sizeof(_wordlen_t)));
		startlen = (unsigned int)((_wordlen_t)blockstart - (_wordlen_t)start);
	}

	/* Compute the number of 32- or 64- bit blocks and the remaining number of bytes */
	residual = count % sizeof(_wordlen_t);
	length = count / sizeof(_wordlen_t);
	if (residual != 0) 	{
		residualstart = (unsigned char *)(blockstart + length);
	}

	/* Get a temporary random number for use in the residuals. Failout if retry fails */
	if (startlen > 0) 	{
#ifdef _WIN64
		if ((success = rdrand_64((uint64_t *)&temprand, 1)) != RDRAND_SUCCESS) return success;
#else
		if ((success = rdrand_32((uint32_t *)&temprand, 1)) != RDRAND_SUCCESS) return success;
#endif
	}

	/* populate the starting misaligned block */
	for (i = 0; i<startlen; i++) 	{
		start[i] = (unsigned char)(temprand & 0xff);
		temprand = temprand >> 8;
	}

	/* populate the central aligned block. Fail out if retry fails */

#ifdef _WIN64
	if ((success = rdrand_get_n_64(length, (uint64_t *)(blockstart))) != RDRAND_SUCCESS) return success;
#else
	if ((success = rdrand_get_n_32(length, (uint32_t *)(blockstart))) != RDRAND_SUCCESS) return success;
#endif
	/* populate the final misaligned block */
	if (residual > 0)
	{
#ifdef _WIN64
		if ((success = rdrand_64((uint64_t *)&temprand, 1)) != RDRAND_SUCCESS) return success;
#else
		if ((success = rdrand_32((uint32_t *)&temprand, 1)) != RDRAND_SUCCESS) return success;
#endif

		for (i = 0; i<residual; i++) 		{
			residualstart[i] = (unsigned char)(temprand & 0xff);
			temprand = temprand >> 8;
		}
	}
	return RDRAND_SUCCESS;
}

__global__ void ASCON6_eprint(bit64 IV[], bit64 key[], bit64 nonce[], __int64 counter[]) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;

	initial0 = IV[threadIndex];
	initial1 = key[2 * threadIndex];
	initial2 = key[2 * threadIndex + 1];
	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];

	for (int c = 0; c < TRIALS; c++) {
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2;
		//	pair3 = initial3 ^ 0x0040000000000000;
		//	pair4 = initial4 ^ 0x0040000000000000;
		pair3 = initial3 ^ 0x8000000000000000;
		pair4 = initial4 ^ 0x8000000000000000;

		for (int i = 0; i < 5; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
		t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
		initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
		initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
		for (int i = 0; i < 5; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
		pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
		t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
		pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
		pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
		/*		t1 = 0;
		t0 = initial0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		t0 = pair0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		if (t1 == 0) counter[threadIndex]++;*/

		//	t0 = (initial0 ^ pair0) & 0x9224b6d24b6eda49;
		//		t0 = (initial0 ^ pair0) & 0x0000000000000200;
//		t0 = (initial0 ^ pair0) & 0x9324496da496ddb4; // Authors 6-round
		t0 = (initial0 ^ pair0) & 0x0200000000000000;
//		t0 = (initial0 ^ pair0) & 0x892db492dbb69264;
//		t0 = (initial4 ^ pair4) & 0x297cc63cdc4b8fec;
		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x1111111111111111UL) * 0x1111111111111111UL;
		t0 = (t0 >> 60) & 1;
		if (t0 == 0) counter[threadIndex]++;
	}
}
__global__ void ASCON12_benchmark(bit64 IV[], bit64 key[], bit64 nonce[], bit64 keystream, __int64 trial) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 t0, t1, t2, t3, t4;
	bit64 IV2 = IV[threadIndex];
	bit64 key0 = key[2 * threadIndex];
	bit64 key1 = key[2 * threadIndex]+1;
	bit64 nonce0 = key[2 * threadIndex];
	bit64 nonce1 = key[2 * threadIndex] + 1;
	for (__int64 c = 0; c < trial; c++) {
		initial0 = IV2;
		initial1 = key0;
		initial2 = key1;
		initial3 = nonce0;
		initial4 = nonce1;
#pragma unroll
		for (int i = 0; i < 12; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		nonce1++;
		if (initial0 == keystream) printf("Hello world\n");
	}
}
__global__ void ASCON_crypto24(bit64 IV[], bit64 key[], bit64 nonce[], __int64 counter[], __int64 trial) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;
	initial0 = IV[threadIndex];
	initial1 = key[2 * threadIndex];
	initial2 = key[2 * threadIndex + 1];
	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];
	for (__int64 c = 0; c < trial; c++) {
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2;
		pair3 = initial3 ^ 0x8000000000000000;
		pair4 = initial4 ^ 0x8000000000000000;
#pragma unroll
		for (int i = 0; i < 5; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);			t1 = rotater(initial0, 28);			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);			t1 = rotater(initial1, 39);			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);			t1 = rotater(initial2, 6);			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);			t1 = rotater(initial3, 17);			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);			t1 = rotater(initial4, 41);			initial4 ^= t0 ^ t1;
		}
		initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
		t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
		initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
		initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
#pragma unroll
		for (int i = 0; i < 5; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);			t1 = rotater(pair0, 28);			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);			t1 = rotater(pair1, 39);			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);				t1 = rotater(pair2, 6);			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);			t1 = rotater(pair3, 17);			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);				t1 = rotater(pair4, 41);			pair4 ^= t0 ^ t1;
		}
		pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
		t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
		pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
		pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
		t0 = (initial0 ^ pair0) & 0x0200000000000000;
		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x1111111111111111UL) * 0x1111111111111111UL;
		t0 = (t0 >> 60) & 1;
		if (t0 == 0) counter[threadIndex]++;
	}
}
void ASCON_benchmark() {
	printf("Trial = 2^18 +  ");
	scanf_s("%d", &trial);
	trial = (__int64)1 << trial;
	float milliseconds = 0;
	hipMalloc((void **)&key_d, 2 * sizeof(bit64));
	hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
	hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
	hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
	rdrand_64(key, 0);
	rdrand_64(key + 1, 0);
	hipEvent_t start, stop;
		for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
		for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
		hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
		hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
		hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
		StartCounter();
		hipDeviceSynchronize(); clock_t beginTime = clock();
		hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start);
		ASCON12_benchmark << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, 0x0123456789abcdef, trial);
		hipEventRecord(stop);	hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);	printf("Time elapsed: %f milliseconds ", milliseconds);	printf("Time of kernel: %lf\n", GetCounter());	
}
void ASCON55_crypto24() {
	FILE* fp;	fopen_s(&fp, "crypto24.txt", "w");
	__int64 *counter_d, total_counter = 0, bias, average_bias = 0, experiment;
	__int64 *counter;
	printf("Trial = 2^18 +  ");
	scanf_s("%d", &trial);
	trial = (__int64)1 << trial;
	experiment = trial*THREADS*BLOCKS;
	float milliseconds = 0;
	total_counter = 0;
	hipMalloc((void **)&key_d, 2 * sizeof(bit64));
	hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
	hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
	hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
	hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
	rdrand_64(key, 0);	rdrand_64(key + 1, 0);
	
	counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
	hipEvent_t start, stop;
	for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
	for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
	hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
	hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
	hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
	hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
	StartCounter();
	hipDeviceSynchronize(); clock_t beginTime = clock();
	hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start);
	ASCON_crypto24 << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, counter_d, trial);
	hipEventRecord(stop);	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);	printf("Time elapsed: %f milliseconds ", milliseconds);	printf("Time of kernel: %lf\n", GetCounter());

	hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
	for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
	bias = (experiment) / 2 - total_counter;
//	printf("Total counter: %I64d Bias: %I64d\n", total_counter, bias);
	printf("\nTotal counter: %I64d\nDifference from the Expected Value: %I64d\nBias: 2^-%lf (For an experiment with 2^%lf data)\n", total_counter, bias, ((log(BLOCKS) + log(THREADS) + log(trial)) / log(2)) - (log(abs(bias)) / log(2)), (log(BLOCKS) + log(THREADS) + log(trial)) / log(2));
	fprintf(fp, "Total counter: %I64d Bias: %I64d\n", total_counter, bias);
	hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
	fclose(fp);
}
int main(void) {
	hipSetDevice(0);
	__int64 *counter_d, total_counter=0, bias, average_bias=0, experiment;

	nonce = (bit64*)calloc(BLOCKS * THREADS * 2, sizeof(bit64));
	keyrows = (bit64*)calloc(BLOCKS * THREADS * 2, sizeof(bit64));
	IV = (bit64*)calloc(BLOCKS * THREADS, sizeof(bit64));
	int choice = 0, key_choice = 0;
	printf(
		"(1) CUDA ASCON 6-round DL Check of CRYPTO'24 Paper (eprint 2024-871)\n"
		"(2) CUDA ASCON 5.5-round DL Verification of CRYPTO'24 Paper  (eprint 2024-871)\n"
		"(3) CUDA ASCON 12-round Initialization Benchmark\n"
		"Choice: "
		);
	scanf_s("%d", &choice);
	if (choice == 1) {
		FILE* fp;
		fopen_s(&fp, "eprint6r.txt", "w");
		printf("Trial = 2^30 +  ");
		scanf_s("%d", &trial);
		//trial = pow(2, trial);
		trial = (__int64)1 << trial;
		experiment = TRIALS*trial*THREADS*BLOCKS;
		for (int m = 0; m < 1; m++) {
			//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			StartCounter();
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
				for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				ASCON6_eprint << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, counter_d);
			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			printf("Time of kernel: %lf\n", GetCounter());
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
//			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			printf("\nTotal counter: %I64d\nDifference from the Expected Value: %I64d\nBias: 2^-%lf (For an experiment with 2^%lf data)\n", total_counter, bias, ((log(BLOCKS) + log(THREADS) + log(TRIALS) + log(trial)) / log(2)) - (log(abs(bias)) / log(2)), (log(BLOCKS) + log(THREADS) + log(trial) + log(TRIALS)) / log(2));
			fprintf(fp,"%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
			fclose(fp);
		}
	}
	else if (choice == 2)	ASCON55_crypto24();
	else if (choice == 3) ASCON_benchmark();
	free(nonce); free(keyrows); free(IV);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
}

