#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <Windows.h>
#include "rdrand.h"
#include <intrin.h>
#include <immintrin.h>
#include <string.h>
#include <ctime>

#define BLOCKS				256
#define THREADS				1024
#define TRIALS				1024*256
//__int64 trial = 1024*4, keys = 100;
__int64 trial = 1, keys = 100;
int rotation = 0;
double PCFreq = 0.0;
__int64 CounterStart = 0;

#define RDRAND_MASK	0x40000000
#define RETRY_LIMIT 10
#ifdef _WIN64
typedef uint64_t _wordlen_t;
#else
typedef uint32_t _wordlen_t;
#endif
#define bit32 unsigned int
#define bit64 unsigned __int64 
bit64 key[2], *key_d, *nonce, *nonce_d, *IV_d, *keyrows_d, *keyrows, *IV;
bit64 state[5] = { 0 }, t[5] = { 0 };
bit64 constants[16] = { 0xf0, 0xe1, 0xd2, 0xc3, 0xb4, 0xa5, 0x96, 0x87, 0x78, 0x69, 0x5a, 0x4b, 0x3c, 0x2d, 0x1e, 0x0f };
int key_choice = 0;
void StartCounter(){
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		printf("QueryPerformanceFrequency failed!\n");

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter(){
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}
void print_state(bit64 state[5]) {	for (int i = 0; i < 5; i++) printf("%016I64x\n", state[i]);}
void add_constant(bit64 state[5], int i, int a) {	state[2] = state[2] ^ constants[12 - a + i];}
void sbox(bit64 x[5]) {
	x[0] ^= x[4]; x[4] ^= x[3]; x[2] ^= x[1];
	t[0] = x[0]; t[1] = x[1]; t[2] = x[2]; t[3] = x[3]; t[4] = x[4];
	t[0] = ~t[0]; t[1] = ~t[1]; t[2] = ~t[2]; t[3] = ~t[3]; t[4] = ~t[4];
	t[0] &= x[1]; t[1] &= x[2]; t[2] &= x[3]; t[3] &= x[4]; t[4] &= x[0];
	x[0] ^= t[1]; x[1] ^= t[2]; x[2] ^= t[3]; x[3] ^= t[4]; x[4] ^= t[0];
	x[1] ^= x[0]; x[0] ^= x[4]; x[3] ^= x[2]; x[2] = ~x[2];
}
bit64 rotate(bit64 x, int l) {
	bit64 temp;
	temp = (x >> l) ^ (x << (64 - l));
	return temp;
}
__device__ bit64 rotater(bit64 x, int l) {
	bit64 temp;
	temp = (x >> l) ^ (x << (64 - l));
	return temp;
}
void linear(bit64 state[5]) {
	bit64 temp0, temp1;
	temp0 = rotate(state[0], 19);
	temp1 = rotate(state[0], 28);
	state[0] ^= temp0 ^ temp1;
	temp0 = rotate(state[1], 61);
	temp1 = rotate(state[1], 39);
	state[1] ^= temp0 ^ temp1;
	temp0 = rotate(state[2], 1);
	temp1 = rotate(state[2], 6);
	state[2] ^= temp0 ^ temp1;
	temp0 = rotate(state[3], 10);
	temp1 = rotate(state[3], 17);
	state[3] ^= temp0 ^ temp1;
	temp0 = rotate(state[4], 7);
	temp1 = rotate(state[4], 41);
	state[4] ^= temp0 ^ temp1;
}
void p(bit64 state[5], int a) {
	for (int i = 0; i < a; i++) {
		add_constant(state, i, a);
		sbox(state);
		linear(state);
	}
}
void initialization(bit64 state[5], bit64 key[2]) {
	p(state, 12);
	state[3] ^= key[0];
	state[4] ^= key[1];
}
void encrypt(bit64 state[5], int length, bit64 plaintext[], bit64 ciphertext[]) {
	ciphertext[0] = plaintext[0] ^ state[0];
	for (int i = 1; i < length; i++) {
		p(state, 6);
		ciphertext[i] = plaintext[i] ^ state[0];
		state[0] = plaintext[i] ^ state[0];
	}
}
void decrypt(bit64 state[5], int length, bit64 plaintext[], bit64 ciphertext[]) {
	ciphertext[0] = plaintext[0] ^ state[0];
	for (int i = 1; i < length; i++) {
		p(state, 6);
		ciphertext[i] = plaintext[i] ^ state[0];
		state[0] = plaintext[i];
	}
}
void main_old() {
	bit64 IV = 0x80400c0600000000, key[2] = { 0xffffffffffffffff, 0x123456789abcdef0 }, nonce[2] = { 0 };
	bit64 plaintext[10] = { 0 }, ciphertext[10];
	state[0] = IV;
	state[1] = key[0];
	state[2] = key[1];
	state[3] = nonce[0];
	state[4] = nonce[1];
	print_state(state); printf("\n");
	//p(state, 12);
	initialization(state, key);
	print_state(state);
	encrypt(state, 10, plaintext, ciphertext); printf("\n");
	for (int i = 0; i < 10; i++) printf("%016I64x\n", ciphertext[i]);
	state[0] = IV;
	state[1] = key[0];
	state[2] = key[1];
	state[3] = nonce[0];
	state[4] = nonce[1]; printf("\n");
	print_state(state); printf("\n");
	//p(state, 12);
	initialization(state, key);
	print_state(state);
	decrypt(state, 10, ciphertext, plaintext); printf("\n");
	for (int i = 0; i < 10; i++) printf("%016I64x\n", plaintext[i]);
}
/*! \brief Queries cpuid to see if rdrand is supported
*
* rdrand support in a CPU is determined by examining the 30th bit of the ecx
* register after calling cpuid.
*
* \return bool of whether or not rdrand is supported
*/
int RdRand_cpuid() {
	int info[4] = { -1, -1, -1, -1 };
	/* Are we on an Intel processor? */
	__cpuid(info, /*feature bits*/0);
	if (memcmp((void *)&info[1], (void *) "Genu", 4) != 0 ||
		memcmp((void *)&info[3], (void *) "ineI", 4) != 0 ||
		memcmp((void *)&info[2], (void *) "ntel", 4) != 0) {
		return 0;
	}
	/* Do we have RDRAND? */
	__cpuid(info, /*feature bits*/1);
	int ecx = info[2];
	if ((ecx & RDRAND_MASK) == RDRAND_MASK)
		return 1;
	else
		return 0;
}
/*! \brief Determines whether or not rdrand is supported by the CPU
*
* This function simply serves as a cache of the result provided by cpuid,
* since calling cpuid is so expensive. The result is stored in a static
* variable to save from calling cpuid on each invocation of rdrand.
*
* \return bool/int of whether or not rdrand is supported
*/
int RdRand_isSupported() {
	static int supported = RDRAND_SUPPORT_UNKNOWN;

	if (supported == RDRAND_SUPPORT_UNKNOWN)
	{
		if (RdRand_cpuid())
			supported = RDRAND_SUPPORTED;
		else
			supported = RDRAND_UNSUPPORTED;
	}

	return (supported == RDRAND_SUPPORTED) ? 1 : 0;
}
int rdrand_16(uint16_t* x, int retry) {
	if (RdRand_isSupported())	{
		if (retry)		{
			for (int i = 0; i < RETRY_LIMIT; i++)			{
				if (_rdrand16_step(x))
					return RDRAND_SUCCESS;
			}
			return RDRAND_NOT_READY;
		}
		else		{
			if (_rdrand16_step(x))				return RDRAND_SUCCESS;
			else				return RDRAND_NOT_READY;
		}
	}
	else	{		return RDRAND_UNSUPPORTED;	}
}
int rdrand_32(uint32_t* x, int retry) {
	if (RdRand_isSupported())	{
		if (retry)		{
			for (int i = 0; i < RETRY_LIMIT; i++)			{
				if (_rdrand32_step(x))
					return RDRAND_SUCCESS;
			}
			return RDRAND_NOT_READY;
		}
		else		{
			if (_rdrand32_step(x))
				return RDRAND_SUCCESS;
			else
				return RDRAND_NOT_READY;
		}
	}
	else	{		return RDRAND_UNSUPPORTED;	}
}
int rdrand_64(uint64_t* x, int retry) {
	if (RdRand_isSupported())	{
		if (retry)		{
			for (int i = 0; i < RETRY_LIMIT; i++)			{
				if (_rdrand64_step(x))
					return RDRAND_SUCCESS;
			}
			return RDRAND_NOT_READY;
		}
		else		{
			if (_rdrand64_step(x))				return RDRAND_SUCCESS;
			else				return RDRAND_NOT_READY;
		}
	}
	else	{		return RDRAND_UNSUPPORTED;	}
}
int rdrand_get_n_64(unsigned int n, uint64_t *dest) {
	int success;
	int count;
	unsigned int i;

	for (i = 0; i<n; i++) 	{
		count = 0;
		do 		{
			success = rdrand_64(dest, 1);
		} while ((success == 0) && (count++ < RETRY_LIMIT));
		if (success != RDRAND_SUCCESS) return success;
		dest = &(dest[1]);
	}
	return RDRAND_SUCCESS;
}
int rdrand_get_n_32(unsigned int n, uint32_t *dest) {
	int success;
	int count;
	unsigned int i;
	for (i = 0; i<n; i++) 	{
		count = 0;
		do 		{
			success = rdrand_32(dest, 1);
		} while ((success == 0) && (count++ < RETRY_LIMIT));
		if (success != RDRAND_SUCCESS) return success;
		dest = &(dest[1]);
	}
	return RDRAND_SUCCESS;
}
int rdrand_get_bytes(unsigned int n, unsigned char *dest) {
	unsigned char *start;
	unsigned char *residualstart;
	_wordlen_t *blockstart;
	_wordlen_t i, temprand;
	unsigned int count;
	unsigned int residual;
	unsigned int startlen;
	unsigned int length;
	int success;

	/* Compute the address of the first 32- or 64- bit aligned block in the destination buffer, depending on whether we are in 32- or 64-bit mode */
	start = dest;
	if (((uint32_t)start % (uint32_t) sizeof(_wordlen_t)) == 0) 	{
		blockstart = (_wordlen_t *)start;
		count = n;
		startlen = 0;
	}
	else 	{
		blockstart = (_wordlen_t *)(((_wordlen_t)start & ~(_wordlen_t)(sizeof(_wordlen_t)-1)) + (_wordlen_t)sizeof(_wordlen_t));
		count = n - (sizeof(_wordlen_t)-(unsigned int)((_wordlen_t)start % sizeof(_wordlen_t)));
		startlen = (unsigned int)((_wordlen_t)blockstart - (_wordlen_t)start);
	}

	/* Compute the number of 32- or 64- bit blocks and the remaining number of bytes */
	residual = count % sizeof(_wordlen_t);
	length = count / sizeof(_wordlen_t);
	if (residual != 0) 	{
		residualstart = (unsigned char *)(blockstart + length);
	}

	/* Get a temporary random number for use in the residuals. Failout if retry fails */
	if (startlen > 0) 	{
#ifdef _WIN64
		if ((success = rdrand_64((uint64_t *)&temprand, 1)) != RDRAND_SUCCESS) return success;
#else
		if ((success = rdrand_32((uint32_t *)&temprand, 1)) != RDRAND_SUCCESS) return success;
#endif
	}

	/* populate the starting misaligned block */
	for (i = 0; i<startlen; i++) 	{
		start[i] = (unsigned char)(temprand & 0xff);
		temprand = temprand >> 8;
	}

	/* populate the central aligned block. Fail out if retry fails */

#ifdef _WIN64
	if ((success = rdrand_get_n_64(length, (uint64_t *)(blockstart))) != RDRAND_SUCCESS) return success;
#else
	if ((success = rdrand_get_n_32(length, (uint32_t *)(blockstart))) != RDRAND_SUCCESS) return success;
#endif
	/* populate the final misaligned block */
	if (residual > 0)
	{
#ifdef _WIN64
		if ((success = rdrand_64((uint64_t *)&temprand, 1)) != RDRAND_SUCCESS) return success;
#else
		if ((success = rdrand_32((uint32_t *)&temprand, 1)) != RDRAND_SUCCESS) return success;
#endif

		for (i = 0; i<residual; i++) 		{
			residualstart[i] = (unsigned char)(temprand & 0xff);
			temprand = temprand >> 8;
		}
	}
	return RDRAND_SUCCESS;
}
void generate_data() {
	FILE *fp;
	int count = 0, multiplier = 0;
	int i, m, j;
	uint64_t d[10] = { 0 };
	printf("Enter number of sequences to be generated: ");
	scanf_s("%d", &count);
	printf("Enter n x 64 (concatenation count, 1,2,3,4,5,6,8,12,16,24,32,48,64): ");
	scanf_s("%d", &multiplier);

	if (multiplier == 1) fopen_s(&fp, "INTEL64.txt", "w");
	else if (multiplier == 2) fopen_s(&fp, "INTEL128.txt", "w");
	else if (multiplier == 3) fopen_s(&fp, "INTEL192.txt", "w");
	else if (multiplier == 4) fopen_s(&fp, "INTEL256.txt", "w");
	else if (multiplier == 6) fopen_s(&fp, "INTEL384.txt", "w");
	else if (multiplier == 8) fopen_s(&fp, "INTEL512.txt", "w");
	else if (multiplier == 12) fopen_s(&fp, "INTEL768.txt", "w");
	else if (multiplier == 16) fopen_s(&fp, "INTEL1024.txt", "w");
	else if (multiplier == 24) fopen_s(&fp, "INTEL1536.txt", "w");
	else if (multiplier == 32) fopen_s(&fp, "INTEL2048.txt", "w");
	else if (multiplier == 48) fopen_s(&fp, "INTEL3072.txt", "w");
	else if (multiplier == 64) fopen_s(&fp, "INTEL4096.txt", "w");
	else fopen_s(&fp, "INTEL.txt", "w");
	for (i = 0; i < count; i++) {
		for (m = 0; m < multiplier; m++) {
			rdrand_64(d, 0);
			for (j = 0; j < 64; j++) {
				if ((d[0] >> j) & 0x1) fprintf(fp, "1");
				else fprintf(fp, "0");
			}
		}
		fprintf(fp, "\n");
	}
	fclose(fp);
}
void test_intel() {
	uint64_t test = 0;
	uint64_t b[65536] = { 0 }, maximum = 0, minimum = 0, minpos = 0, maxpos = 0;
	uint16_t c[10] = { 0 };
	printf("Test count is n x 65536. Enter expected value n: ");
	scanf_s("%I64d", &test);
	minimum = test;
	test = test * 65536;
	printf("Loop: %I64d\n", test);
	for (uint64_t i = 0; i < test; i++) {
		rdrand_16(c, 0);
		//		if (i%1024==0) b[c[0]]++;
		b[c[0]]++;
	}
	for (int i = 0; i < 65536; i++) {
		if (b[i] > maximum) { maximum = b[i]; maxpos = i; }
		if (b[i] < minimum) { minimum = b[i]; minpos = i; }
	}
	printf("Maximum = %I64d (at %I64d)\n", maximum, maxpos);
	printf("Minimum = %I64d (at %I64d)\n", minimum, minpos);
}
void test_periodicity() {
	uint64_t target = 0, c[10] = { 0 };
	int periodicity;
	rdrand_64(c, 0);
	target = c[0];
	for (periodicity = 1; periodicity < 0x80000000; periodicity = periodicity << 1) {
		printf("Periodicity: %I64d\n", periodicity);
		for (uint64_t i = 0; i < 0x100000000; i++) {
			rdrand_64(c, periodicity);
			if (c[0] == target) {
				printf("PERIOD: %I64x, offset: %I64d\n", periodicity, i);
				system("PAUSE");
			}
		}
	}
}
void ascon_experiment() {
	bit64 d[2] = { 0 };
	bit64 state[5], state2[5], temp, temp2, initial[5];
	__int64 counter = 0, trial = 1024 * 1024, bias, overall_bias = 0, keys = 100;
	for (int c = 0; c < keys; c++) {
		counter = 0;
		//		rdrand_64(d, 0);	initial[0] = d[0];
		initial[0] = 0x80400c0600000000;
		rdrand_64(d, 0);	initial[1] = d[0];
		rdrand_64(d, 0);	initial[2] = d[0];
		for (int j = 0; j < trial; j++) {
			for (int i = 0; i < 3; i++) state[i] = initial[i];
			for (int i = 3; i < 5; i++){
				rdrand_64(d, 0);
				state[i] = d[0];
			}
			for (int i = 0; i < 5; i++) state2[i] = state[i];
			state2[3] ^= 0x0040000000000000;
			state2[4] ^= 0x0040000000000000;
			//	print_state(state); print_state(state2);
			p(state, 4); p(state2, 4);
			/*		temp = state[0] ^ state2[0];
			temp &= 0x9224b6d24b6eda49;
			temp2 = 0;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);*/
			temp2 = 0;
			temp = state[0] & 0x9224b6d24b6eda49;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
			temp = state2[0] & 0x9224b6d24b6eda49;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
			if (temp2 == 0) counter++;
		}
		bias = trial / 2 - counter;
		printf("Counter: %I64d\n", counter);
		printf("Bias: %I64d\n", bias);
		overall_bias += bias;
	}
	printf("Overall bias: %I64d\n", overall_bias / keys);
}
void ascon_4round_key_recovery() {
	bit64 d[2] = { 0 };
	bit64 state[5], state2[5], temp, temp2, initial[5];
	__int64 counter = 0, trial = 1024 * 1024, bias, overall_bias = 0, keys = 100;
	for (int c = 0; c < keys; c++) {
		counter = 0;
		//		rdrand_64(d, 0);	initial[0] = d[0];
		initial[0] = 0x80400c0600000000;
		rdrand_64(d, 0);	d[0] = d[0] & 0xFFBFFFFFFFFFFFFF; if (key_choice == 2 || key_choice == 4) d[0] = d[0] ^ 0x0040000000000000; initial[1] = d[0]; //0xFFBFFFFFFFFFFFFF;
		rdrand_64(d, 0);	d[0] = d[0] & 0xFFBFFFFFFFFFFFFF; if (key_choice == 3 || key_choice == 4) d[0] = d[0] ^ 0x0040000000000000; initial[2] = d[0];
		for (int j = 0; j < trial; j++) {
			for (int i = 0; i < 3; i++) state[i] = initial[i];
			for (int i = 3; i < 5; i++){
				rdrand_64(d, 0);
				state[i] = d[0];
			}
			for (int i = 0; i < 5; i++) state2[i] = state[i];
			state2[3] ^= 0x0040000000000000;
			state2[4] ^= 0x0040000000000000;
			//	print_state(state); print_state(state2);
			p(state, 4); p(state2, 4);
			/*		temp = state[0] ^ state2[0];
			temp &= 0x9224b6d24b6eda49;
			temp2 = 0;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);*/
/*			temp2 = 0;
			temp = state[0] & 0x9224b6d24b6eda49;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
			temp = state2[0] & 0x9224b6d24b6eda49;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
			if (temp2 == 0) counter++;*/
			// Speed up trial
			temp2 = 0;
			temp = state[0] & 0x9224b6d24b6eda49;
			temp = temp ^ (state2[0] & 0x9224b6d24b6eda49);
			for (int i = 0; i < 64; i++) temp2 ^= (temp >> i);
			if ((temp2 &0x1) == 0) counter++;
		}
		bias = trial / 2 - counter;
		printf("Counter: %I64d\n", counter);
		printf("Bias: %I64d\n", bias);
		overall_bias += bias;
	}
	printf("Overall bias: %I64d\n", overall_bias / keys);
}
void ascon_experiment_5round() {
	bit64 d[2] = { 0 };
	bit64 state[5], state2[5], temp, temp2, initial[5];
	__int64 counter = 0, trial = 549755813888, bias, overall_bias = 0, keys = 5; //549755813888
	for (int c = 0; c < keys; c++) {
		counter = 0;
		//		rdrand_64(d, 0);	initial[0] = d[0];
		initial[0] = 0x80400c0600000000;
		rdrand_64(d, 0);	initial[1] = d[0];
		rdrand_64(d, 0);	initial[2] = d[0];
		for (__int64 j = 0; j < trial; j++) {
			for (int i = 0; i < 3; i++) state[i] = initial[i];
			for (int i = 3; i < 5; i++){
				rdrand_64(d, 0);
				state[i] = d[0];
			}
			for (int i = 0; i < 5; i++) state2[i] = state[i];
			state2[3] ^= 0x0000000000020000;
			state2[4] ^= 0x0000000000020000;
			//	print_state(state); print_state(state2);
			p(state, 5); p(state2, 5);
			/*		temp = state[0] ^ state2[0];
			temp &= 0x9224b6d24b6eda49;
			temp2 = 0;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);*/
			temp2 = 0;
			// 0100 1000 1001 0010 1101 1011 0100 1001 0010 1101 1011 1011 0110 1001 0010 0110
			temp = state[0] & 0x4892db492dbb6926;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
			temp = state2[0] & 0x4892db492dbb6926;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
			if (temp2 == 0) counter++;
			if (j % 1073741824 == 0){
				bias = j / 2 - counter;
				printf("j: %d, bias: %I64d\n", j / (1024 * 1024 * 64), bias);
			}
		}
		bias = trial / 2 - counter;
		printf("Counter: %I64d\n", counter);
		printf("Bias: %I64d\n", bias);
		overall_bias += bias;
	}
	printf("Overall bias: %I64d\n", overall_bias / keys);
}
void ascon_5round_key_recovery() {
	bit64 d[2] = { 0 };
	bit64 state[5], state2[5], temp, temp2, initial[5];
	__int64 counter = 0, trial = 34359738368, bias, overall_bias = 0, keys = 5; //549755813888
	for (int c = 0; c < keys; c++) {
		counter = 0;
		//		rdrand_64(d, 0);	initial[0] = d[0];
		initial[0] = 0x80400c0600000000;
		rdrand_64(d, 0);	d[0] = d[0] & 0xFFFFFFFFFFFDFFFF; if (key_choice == 2 || key_choice == 4) d[0] = d[0] ^ 0x0000000000020000; initial[1] = d[0];
		rdrand_64(d, 0);	d[0] = d[0] & 0xFFFFFFFFFFFDFFFF; if (key_choice == 3 || key_choice == 4) d[0] = d[0] ^ 0x0000000000020000; initial[2] = d[0];
		for (__int64 j = 0; j < trial; j++) {
			for (int i = 0; i < 3; i++) state[i] = initial[i];
			for (int i = 3; i < 5; i++){
				rdrand_64(d, 0);
				state[i] = d[0];
			}
			for (int i = 0; i < 5; i++) state2[i] = state[i];
			state2[3] ^= 0x0000000000020000;
			state2[4] ^= 0x0000000000020000;
			//	print_state(state); print_state(state2);
			p(state, 5); p(state2, 5);
			/*		temp = state[0] ^ state2[0];
			temp &= 0x9224b6d24b6eda49;
			temp2 = 0;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);*/
			temp2 = 0;
			// 0100 1000 1001 0010 1101 1011 0100 1001 0010 1101 1011 1011 0110 1001 0010 0110
			temp = state[0] & 0x4892db492dbb6926;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
			temp = state2[0] & 0x4892db492dbb6926;
			for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
			if (temp2 == 0) counter++;
			if (j % 1073741824 == 0){
				bias = j / 2 - counter;
				printf("j: %d, bias: %I64d\n", j / (1073741824), bias);
			}
		}
		bias = trial / 2 - counter;
		printf("Counter: %I64d\n", counter);
		printf("Bias: %I64d\n", bias);
		overall_bias += bias;
	}
	printf("Overall bias: %I64d\n", overall_bias / keys);
}
void ascon_experiment_linear() {
	bit64 d[2] = { 0 };
	bit64 state[5], temp, temp2;
	int counter = 0, trial = 1024 * 1024 * 16, bias;
	for (int j = 0; j < trial; j++) {
		for (int i = 0; i < 5; i++){
			rdrand_64(d, 0);
			state[i] = d[0];
		}
		temp2 = 0;
		temp2 ^= ((state[1] >> 10) & 0x1);
		temp2 ^= ((state[1] >> 17) & 0x1);
		temp2 ^= ((state[2] >> 0) & 0x1);
		temp2 ^= ((state[2] >> 10) & 0x1);
		temp2 ^= ((state[2] >> 17) & 0x1);
		temp2 ^= ((state[3] >> 7) & 0x1);
		temp2 ^= ((state[3] >> 41) & 0x1);
		temp2 ^= ((state[4] >> 7) & 0x1);
		temp2 ^= ((state[4] >> 41) & 0x1);
		p(state, 2);
		temp = state[0] & 0x9224b6d24b6eda49;

		for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);
		if (temp2 == 0) counter++;
	}
	bias = trial / 2 - counter;
	printf("Counter: %d\n", counter);
	printf("Bias: %d\n", bias);
}
void ascon_experiment_truncated() {
	bit64 d[2] = { 0 };
	bit64 state[5], state2[5], initial[5];
	__int64 counter = 0, trial = 10;
	counter = 0;
	initial[0] = 0x80400c0600000000;
	rdrand_64(d, 0);	initial[1] = d[0];
	rdrand_64(d, 0);	initial[2] = d[0];
	for (int j = 0; j < trial; j++) {
		for (int i = 0; i < 3; i++) state[i] = initial[i];
		for (int i = 3; i < 5; i++){
			rdrand_64(d, 0);
			state[i] = d[0];
		}
		for (int i = 0; i < 5; i++) state2[i] = state[i];
		state2[0] ^= 0x8000000000000000;
		state2[3] ^= 0x8000000000000000;
		state2[4] ^= 0x8000000000000000;
		//	print_state(state); print_state(state2);
		p(state, 3); p(state2, 3);
		sbox(state); sbox(state2);
		/*		temp = state[0] ^ state2[0];
		temp &= 0x9224b6d24b6eda49;
		temp2 = 0;
		for (int i = 0; i < 64; i++) temp2 ^= ((temp >> i) & 0x1);*/
		for (int i = 0; i < 5; i++) state[i] ^= state2[i];
		for (int i = 0; i < 5; i++) state[i] &= 0x0000000000000200;
		print_state(state); printf("\n");
	}
}
__global__ void ASCON4(bit64 key[], bit64 nonce[], int key_choice, __int64 counter[]) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;
		initial0 = 0x80400c0600000000;
		initial1 = key[0] & 0xFFBFFFFFFFFFFFFF; if (key_choice == 2 || key_choice == 4) initial1 ^= 0x0040000000000000;
		initial2 = key[1] & 0xFFBFFFFFFFFFFFFF; if (key_choice == 3 || key_choice == 4) initial2 ^= 0x0040000000000000;
		initial3 = nonce[2 * threadIndex ];
		initial4 = nonce[2 * threadIndex + 1];
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2;
		pair3 = initial3 ^ 0x0040000000000000;
		pair4 = initial4 ^ 0x0040000000000000;

		for (int i = 0; i < 4; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		for (int i = 0; i < 4; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
/*		t1 = 0;
		t0 = initial0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		t0 = pair0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		if (t1 == 0) counter[threadIndex]++;*/

		t0 = (initial0 ^ pair0) & 0x9224b6d24b6eda49;
		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x1111111111111111UL) * 0x1111111111111111UL;
		t0 = (t0 >> 60) & 1;
		if (t0 == 0) counter[threadIndex]++;
}
__global__ void ASCON4_eprint(bit64 IV[], bit64 key[], bit64 nonce[], __int64 counter[]) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;

	initial0 = IV[threadIndex];
	initial1 = key[2 * threadIndex];
	initial2 = key[2 * threadIndex + 1];
	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];

	for (int c = 0; c < TRIALS; c++) {
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2;
	//	pair3 = initial3 ^ 0x0040000000000000;
	//	pair4 = initial4 ^ 0x0040000000000000;
		pair3 = initial3 ^ 0x8000000000000000;
		pair4 = initial4 ^ 0x8000000000000000;

		for (int i = 0; i < 3; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
		t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
		initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
		initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
		for (int i = 0; i < 3; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
		pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
		t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
		pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
		pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
		/*		t1 = 0;
		t0 = initial0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		t0 = pair0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		if (t1 == 0) counter[threadIndex]++;*/

		//	t0 = (initial0 ^ pair0) & 0x9224b6d24b6eda49;
		t0 = (initial0 ^ pair0) & 0x0000000000000200;
//		t0 = (initial0 ^ pair0) & 0x0000000000000040;

		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x1111111111111111UL) * 0x1111111111111111UL;
		t0 = (t0 >> 60) & 1;
		if (t0 == 0) counter[threadIndex]++;
	}
}
__global__ void ASCON5(bit64 key[], bit64 nonce[], int key_choice, __int64 counter[]) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0 = 0x80400c0600000000, initial1, initial2, initial3, initial4;
	bit64 pair0 = 0x80400c0600000000, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;
	initial1 = key[0] & 0xFFFFFFFFFFFDFFFF; if (key_choice == 2 || key_choice == 4) initial1 ^= 0x0000000000020000;
	initial2 = key[1] & 0xFFFFFFFFFFFDFFFF; if (key_choice == 3 || key_choice == 4) initial2 ^= 0x0000000000020000;
	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];
	pair1 = initial1;
	pair2 = initial2;
	pair3 = initial3 ^ 0x0000000000020000;
	pair4 = initial4 ^ 0x0000000000020000;

	for (int i = 0; i < 5; i++) {
		initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
		t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
		initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
		initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
		// Liner layer //
		t0 = rotater(initial0, 19);
		t1 = rotater(initial0, 28);
		initial0 ^= t0 ^ t1;
		t0 = rotater(initial1, 61);
		t1 = rotater(initial1, 39);
		initial1 ^= t0 ^ t1;
		t0 = rotater(initial2, 1);
		t1 = rotater(initial2, 6);
		initial2 ^= t0 ^ t1;
		t0 = rotater(initial3, 10);
		t1 = rotater(initial3, 17);
		initial3 ^= t0 ^ t1;
		t0 = rotater(initial4, 7);
		t1 = rotater(initial4, 41);
		initial4 ^= t0 ^ t1;
	}
	for (int i = 0; i < 5; i++) {
		pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
		t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
		pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
		pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
		// Liner layer //
		t0 = rotater(pair0, 19);
		t1 = rotater(pair0, 28);
		pair0 ^= t0 ^ t1;
		t0 = rotater(pair1, 61);
		t1 = rotater(pair1, 39);
		pair1 ^= t0 ^ t1;
		t0 = rotater(pair2, 1);
		t1 = rotater(pair2, 6);
		pair2 ^= t0 ^ t1;
		t0 = rotater(pair3, 10);
		t1 = rotater(pair3, 17);
		pair3 ^= t0 ^ t1;
		t0 = rotater(pair4, 7);
		t1 = rotater(pair4, 41);
		pair4 ^= t0 ^ t1;
	}
	t1 = 0;
	t0 = initial0 & 0x4892db492dbb6926;
	for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
	t0 = pair0 & 0x4892db492dbb6926;
	for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
	if (t1 == 0) counter[threadIndex]++;
}
__global__ void ASCON5b(bit64 key[], bit64 nonce[], int key_choice, __int64 counter[]) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0 = 0x80400c0600000000, initial1, initial2, initial3, initial4;
	bit64 pair0 = 0x80400c0600000000, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;

	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];
	for (int c = 0; c < TRIALS; c++) {
		initial0 = 0x80400c0600000000;
		pair0 = 0x80400c0600000000;
		initial1 = key[0] & 0xFFFFFFFFFFFDFFFF; if (key_choice == 2 || key_choice == 4) initial1 ^= 0x0000000000020000;
		initial2 = key[1] & 0xFFFFFFFFFFFDFFFF; if (key_choice == 3 || key_choice == 4) initial2 ^= 0x0000000000020000;
		pair1 = initial1;
		pair2 = initial2;
		pair3 = initial3 ^ 0x0000000000020000;
		pair4 = initial4 ^ 0x0000000000020000;

		for (int i = 0; i < 5; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		for (int i = 0; i < 5; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
		t1 = 0;
		t0 = initial0 & 0x4892db492dbb6926;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		t0 = pair0 & 0x4892db492dbb6926;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		if (t1 == 0) counter[threadIndex]++;
		initial3 += initial1;
		initial4 += initial2;
	}
}
__global__ void ASCON5c(bit64 key[], bit64 nonce[], int key_choice, __int64 counter[], int rotation) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0 = 0x80400c0600000000, initial1, initial2, initial3, initial4;
	bit64 pair0 = 0x80400c0600000000, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;

	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];
	for (int c = 0; c < TRIALS; c++) {
		t0 = ((bit64)0xFFFFFFFFFFFFF7FF >> rotation) ^ ((bit64)0xFFFFFFFFFFFFF7FF << (64 - rotation));
		t1 = ((bit64)0x0000000000000800 >> rotation) ^ ((bit64)0x0000000000000800 << (64 - rotation));
		initial0 = 0x80400c0600000000;
		pair0 = 0x80400c0600000000;
		initial1 = key[0] & t0; if (key_choice == 2 || key_choice == 4) initial1 ^= t1; // 0xFFFFFFFFFFFEFFFF
		initial2 = key[1] & t0; if (key_choice == 3 || key_choice == 4) initial2 ^= t1;
		pair1 = initial1;
		pair2 = initial2;
		pair3 = initial3 ^ t1;
		pair4 = initial4 ^ t1;

		for (int i = 0; i < 5; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		for (int i = 0; i < 5; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
		t1 = 0;
		t0 = initial0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		t0 = pair0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		if (t1 == 0) counter[threadIndex]++;
		initial3 += initial1;
		initial4 += initial2;
	}
}
__global__ void ASCON6(bit64 key[], bit64 nonce[], int key_choice, __int64 counter[], int rotation) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0 = 0x80400c0600000000, initial1, initial2, initial3, initial4;
	bit64 pair0 = 0x80400c0600000000, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4, v;

	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];
	for (int c = 0; c < TRIALS; c++) {
		t0 = ((bit64)0xFFFFFFFFFFFFF7FF >> rotation) ^ ((bit64)0xFFFFFFFFFFFFF7FF << (64 - rotation));
		t1 = ((bit64)0x0000000000000800 >> rotation) ^ ((bit64)0x0000000000000800 << (64 - rotation));
		initial0 = 0x80400c0600000000;
		pair0 = 0x80400c0600000000;
		initial1 = key[0] & t0; if (key_choice == 2 || key_choice == 4) initial1 ^= t1; // 0xFFFFFFFFFFFEFFFF
		initial2 = key[1] & t0; if (key_choice == 3 || key_choice == 4) initial2 ^= t1;
		pair1 = initial1;
		pair2 = initial2;
		pair3 = initial3 ^ t1;
		pair4 = initial4 ^ t1;

		for (int i = 0; i < 6; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		for (int i = 0; i < 6; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
/*		t1 = 0;
		t0 = initial0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		t0 = pair0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		if (t1 == 0) counter[threadIndex]++;*/
		// Speed up trial
/*		t1 = 0;
		t0 = initial0 & 0x9224b6d24b6eda49;
		t0 = t0 ^ (pair0 & 0x9224b6d24b6eda49);
		for (int i = 0; i < 64; i++) t1 ^= (t0 >> i);
		if ((t1 & 0x1) == 0) counter[threadIndex]++;*/

		v = (initial0 ^ pair0) & 0x9224b6d24b6eda49;
		v ^= v >> 1;
		v ^= v >> 2;
		v = (v & 0x1111111111111111UL) * 0x1111111111111111UL;
		v = (v >> 60) & 1;
		if (v == 0) counter[threadIndex]++;
		initial3 += initial1;
		initial4 += initial2;
	}
}
__global__ void ASCON6_eprint(bit64 IV[], bit64 key[], bit64 nonce[], __int64 counter[]) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;

	initial0 = IV[threadIndex];
	initial1 = key[2 * threadIndex];
	initial2 = key[2 * threadIndex + 1];
	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];

	for (int c = 0; c < TRIALS; c++) {
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2;
		//	pair3 = initial3 ^ 0x0040000000000000;
		//	pair4 = initial4 ^ 0x0040000000000000;
		pair3 = initial3 ^ 0x8000000000000000;
		pair4 = initial4 ^ 0x8000000000000000;

		for (int i = 0; i < 5; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
		t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
		initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
		initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
		for (int i = 0; i < 5; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
		pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
		t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
		pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
		pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
		/*		t1 = 0;
		t0 = initial0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		t0 = pair0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		if (t1 == 0) counter[threadIndex]++;*/

		//	t0 = (initial0 ^ pair0) & 0x9224b6d24b6eda49;
		//		t0 = (initial0 ^ pair0) & 0x0000000000000200;
//		t0 = (initial0 ^ pair0) & 0x9324496da496ddb4; // Authors 6-round
		t0 = (initial0 ^ pair0) & 0x0200000000000000;
//		t0 = (initial0 ^ pair0) & 0x892db492dbb69264;
//		t0 = (initial4 ^ pair4) & 0x297cc63cdc4b8fec;
		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x1111111111111111UL) * 0x1111111111111111UL;
		t0 = (t0 >> 60) & 1;
		if (t0 == 0) counter[threadIndex]++;
	}
}
__global__ void ASCON1_eprint(bit64 IV[], bit64 key[], bit64 nonce[], __int64 counter[]) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;

	initial0 = IV[threadIndex];
	initial1 = key[2 * threadIndex];
	initial2 = key[2 * threadIndex + 1];
	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];

	for (int c = 0; c < TRIALS; c++) {
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2;
		//	pair3 = initial3 ^ 0x0040000000000000;
		//	pair4 = initial4 ^ 0x0040000000000000;
		pair3 = initial3 ^ 0x8000000000000000;
		pair4 = initial4 ^ 0x8000000000000000;

		for (int i = 0; i < 1; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
		t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
		initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
		initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
		for (int i = 0; i < 1; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
		pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
		t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
		pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
		pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
		/*		t1 = 0;
		t0 = initial0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		t0 = pair0 & 0x9224b6d24b6eda49;
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);
		if (t1 == 0) counter[threadIndex]++;*/

		//	t0 = (initial0 ^ pair0) & 0x9224b6d24b6eda49;
		//		t0 = (initial0 ^ pair0) & 0x0000000000000200;
		//		t0 = (initial0 ^ pair0) & 0x9324496da496ddb4; // Authors 6-round
		t0 = (initial0 ^ pair0) & 0x0200000000000000;
		//		t0 = (initial0 ^ pair0) & 0x892db492dbb69264;
		//		t0 = (initial4 ^ pair4) & 0x297cc63cdc4b8fec;
		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x1111111111111111UL) * 0x1111111111111111UL;
		t0 = (t0 >> 60) & 1;
		if (t0 == 0) counter[threadIndex]++;
	}
}
__global__ void ASCON2_hossein(bit64 IV[], bit64 key[], bit64 nonce[], __int64 counter[]) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;

	initial0 = IV[threadIndex];
	initial1 = key[2 * threadIndex];
	initial2 = key[2 * threadIndex + 1];
	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];

	for (int c = 0; c < TRIALS; c++) {
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2 ^ 0x0000000a00800000;
		pair3 = initial3 ^ 0x0000000c230e1000;
		pair4 = initial4;
/*		input diff.middle:
		input_diff.x[0] = 0x0000000000000000;
		input_diff.x[1] = 0x0000000000000000;
		input_diff.x[2] = 0x0000000a00800000;
		input_diff.x[3] = 0x0000000c230e1000;
		input_diff.x[4] = 0x0000000000000000;*/

		for (int i = 0; i < 2; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		for (int i = 0; i < 2; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);
			t1 = rotater(pair0, 28);
			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);
			t1 = rotater(pair1, 39);
			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);
			t1 = rotater(pair2, 6);
			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);
			t1 = rotater(pair3, 17);
			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);
			t1 = rotater(pair4, 41);
			pair4 ^= t0 ^ t1;
		}
/*		output mask middle :
		output_mask.x[0] = 0x0000000000000000;
		output_mask.x[1] = 0x2180000000000000;
		output_mask.x[2] = 0x2180000000000000;
		output_mask.x[3] = 0x0000000000000000;
		output_mask.x[4] = 0x0000000000000000;*/
		t0 = (initial1 ^ pair1 ^ initial2 ^ pair2) & 0x2180000000000000;
		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x1111111111111111UL) * 0x1111111111111111UL;
		t0 = (t0 >> 60) & 1;
		if (t0 == 0) counter[threadIndex]++;
	}
}
__global__ void ASCON12_benchmark(bit64 IV[], bit64 key[], bit64 nonce[], bit64 keystream, __int64 trial) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 t0, t1, t2, t3, t4;
	bit64 IV2 = IV[threadIndex];
	bit64 key0 = key[2 * threadIndex];
	bit64 key1 = key[2 * threadIndex]+1;
	bit64 nonce0 = key[2 * threadIndex];
	bit64 nonce1 = key[2 * threadIndex] + 1;
	for (__int64 c = 0; c < trial; c++) {
		initial0 = IV2;
		initial1 = key0;
		initial2 = key1;
		initial3 = nonce0;
		initial4 = nonce1;
#pragma unroll
		for (int i = 0; i < 12; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);
			t1 = rotater(initial0, 28);
			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);
			t1 = rotater(initial1, 39);
			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);
			t1 = rotater(initial2, 6);
			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);
			t1 = rotater(initial3, 17);
			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);
			t1 = rotater(initial4, 41);
			initial4 ^= t0 ^ t1;
		}
		nonce1++;
		if (initial0 == keystream) printf("Hello world\n");
	}
}
__global__ void ASCON_crypto24(bit64 IV[], bit64 key[], bit64 nonce[], __int64 counter[], __int64 trial) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4;
	initial0 = IV[threadIndex];
	initial1 = key[2 * threadIndex];
	initial2 = key[2 * threadIndex + 1];
	initial3 = nonce[2 * threadIndex];
	initial4 = nonce[2 * threadIndex + 1];
	for (__int64 c = 0; c < trial; c++) {
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2;
		pair3 = initial3 ^ 0x8000000000000000;
		pair4 = initial4 ^ 0x8000000000000000;
#pragma unroll
		for (int i = 0; i < 5; i++) {
			initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
			t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
			initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
			initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
			// Liner layer //
			t0 = rotater(initial0, 19);			t1 = rotater(initial0, 28);			initial0 ^= t0 ^ t1;
			t0 = rotater(initial1, 61);			t1 = rotater(initial1, 39);			initial1 ^= t0 ^ t1;
			t0 = rotater(initial2, 1);			t1 = rotater(initial2, 6);			initial2 ^= t0 ^ t1;
			t0 = rotater(initial3, 10);			t1 = rotater(initial3, 17);			initial3 ^= t0 ^ t1;
			t0 = rotater(initial4, 7);			t1 = rotater(initial4, 41);			initial4 ^= t0 ^ t1;
		}
		initial0 ^= initial4; initial4 ^= initial3; initial2 ^= initial1;
		t0 = initial0; t1 = initial1; t2 = initial2; t3 = initial3; t4 = initial4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= initial1; t1 &= initial2; t2 &= initial3; t3 &= initial4; t4 &= initial0;
		initial0 ^= t1; initial1 ^= t2; initial2 ^= t3; initial3 ^= t4; initial4 ^= t0;
		initial1 ^= initial0; initial0 ^= initial4; initial3 ^= initial2; initial2 = ~initial2;
#pragma unroll
		for (int i = 0; i < 5; i++) {
			pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
			t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
			t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
			t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
			pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
			pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
			// Liner layer //
			t0 = rotater(pair0, 19);			t1 = rotater(pair0, 28);			pair0 ^= t0 ^ t1;
			t0 = rotater(pair1, 61);			t1 = rotater(pair1, 39);			pair1 ^= t0 ^ t1;
			t0 = rotater(pair2, 1);				t1 = rotater(pair2, 6);			pair2 ^= t0 ^ t1;
			t0 = rotater(pair3, 10);			t1 = rotater(pair3, 17);			pair3 ^= t0 ^ t1;
			t0 = rotater(pair4, 7);				t1 = rotater(pair4, 41);			pair4 ^= t0 ^ t1;
		}
		pair0 ^= pair4; pair4 ^= pair3; pair2 ^= pair1;
		t0 = pair0; t1 = pair1; t2 = pair2; t3 = pair3; t4 = pair4;
		t0 = ~t0; t1 = ~t1; t2 = ~t2; t3 = ~t3; t4 = ~t4;
		t0 &= pair1; t1 &= pair2; t2 &= pair3; t3 &= pair4; t4 &= pair0;
		pair0 ^= t1; pair1 ^= t2; pair2 ^= t3; pair3 ^= t4; pair4 ^= t0;
		pair1 ^= pair0; pair0 ^= pair4; pair3 ^= pair2; pair2 = ~pair2;
		t0 = (initial0 ^ pair0) & 0x0200000000000000;
		t0 ^= t0 >> 1;
		t0 ^= t0 >> 2;
		t0 = (t0 & 0x1111111111111111UL) * 0x1111111111111111UL;
		t0 = (t0 >> 60) & 1;
		if (t0 == 0) counter[threadIndex]++;
	}
}
void ASCON_benchmark() {
	printf("Trial = 2^18 +  ");
	scanf_s("%d", &trial);
	trial = (__int64)1 << trial;
	float milliseconds = 0;
	hipMalloc((void **)&key_d, 2 * sizeof(bit64));
	hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
	hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
	hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
	rdrand_64(key, 0);
	rdrand_64(key + 1, 0);
	hipEvent_t start, stop;
		for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
		for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
		hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
		hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
		hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
		StartCounter();
		hipDeviceSynchronize(); clock_t beginTime = clock();
		hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start);
		ASCON12_benchmark << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, 0x0123456789abcdef, trial);
		hipEventRecord(stop);	hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);	printf("Time elapsed: %f milliseconds ", milliseconds);	printf("Time of kernel: %lf\n", GetCounter());
	
}
void ASCON55_crypto24() {
	FILE* fp;	fopen_s(&fp, "crypto24.txt", "w");
	__int64 *counter_d, total_counter = 0, bias, average_bias = 0, experiment;
	__int64 *counter;
	printf("Trial = 2^18 +  ");
	scanf_s("%d", &trial);
	trial = (__int64)1 << trial;
	experiment = trial*THREADS*BLOCKS;
	float milliseconds = 0;
	total_counter = 0;
	hipMalloc((void **)&key_d, 2 * sizeof(bit64));
	hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
	hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
	hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
	hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
	rdrand_64(key, 0);	rdrand_64(key + 1, 0);
	
	counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
	hipEvent_t start, stop;
	for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
	for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
	hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
	hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
	hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
	hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
	StartCounter();
	hipDeviceSynchronize(); clock_t beginTime = clock();
	hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start);
	ASCON_crypto24 << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, counter_d, trial);
	hipEventRecord(stop);	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);	printf("Time elapsed: %f milliseconds ", milliseconds);	printf("Time of kernel: %lf\n", GetCounter());

	hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
	for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
	bias = (experiment) / 2 - total_counter;
	printf("Total counter: %I64d Bias: %I64d\n", total_counter, bias);
	fprintf(fp, "Total counter: %I64d Bias: %I64d\n", total_counter, bias);
	hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
	fclose(fp);
}
int main(void) {
	hipSetDevice(0);
	__int64 *counter_d, total_counter=0, bias, average_bias=0, experiment;

	nonce = (bit64*)calloc(BLOCKS * THREADS * 2, sizeof(bit64));
	keyrows = (bit64*)calloc(BLOCKS * THREADS * 2, sizeof(bit64));
	IV = (bit64*)calloc(BLOCKS * THREADS, sizeof(bit64));
	int choice = 0, key_choice = 0;
	printf("(1) Generate Random bits\n"
		"(2) Test INTEL DRNG\n"
		"(3) Test Periodicity\n"
		"(4) ASCON experiment\n"
		"(5) ASCON experiment linear\n"
		"(6) ASCON experiment truncated\n"
		"(7) ASCON experiment 5-round\n"
		"(8) ASCON 4-round key recovery\n"
		"(9) ASCON 5-round key recovery\n"
		"(10) CUDA ASCON 4-round key recovery\n"
		"(11) CUDA ASCON 5-round key recovery\n"
		"(12) CUDA ASCON 5-round key recovery in kernel\n"
		"(13) CUDA ASCON 5-round key recovery in kernel Trial and Error\n"
		"(14) CUDA ASCON 6-round key recovery in kernel Trial and Error\n"
		"(15) CUDA ASCON 6-round key recovery in kernel Automatic Trial and Error\n"
		"...\n"
		"(20) CUDA ASCON 6-round DL Check (eprint 2024-871)\n"
		"(40) CUDA ASCON 4-round key recovery modified for DL Check (eprint 2024-871)\n"
		"(50) CUDA ASCON 2-round Correlation Check 1 (Hossein)\n"
		"(100) CUDA ASCON 1-round DL Check (eprint 2024-871) Counter sanity check\n"
		"...\n"
		"(200) CUDA ASCON 12-round Initialization Benchmark\n"
		"(201) CUDA ASCON 5.5-round DL Verification of CRYPTO'24 Paper\n"
		"Choice: "
		);
	scanf_s("%d", &choice);
	if (choice == 1) generate_data();
	else if (choice == 2) test_intel();
	else if (choice == 3) test_periodicity();
	else if (choice == 4) ascon_experiment();
	else if (choice == 5) ascon_experiment_linear();
	else if (choice == 6) ascon_experiment_truncated();
	else if (choice == 7) ascon_experiment_5round();
	else if (choice == 8)  {
		printf("Select key (1-4): ");
		scanf_s("%d", &key_choice);
		ascon_4round_key_recovery();
	}
	else if (choice == 9)  {
		printf("Select key (1-4): ");
		scanf_s("%d", &key_choice);
		ascon_5round_key_recovery();
	}
	else if (choice == 10) {
		printf("Select key (1-4): ");
		scanf_s("%d", &key_choice);
		printf("Trial = 2^15 +  ");
		scanf_s("%d", &trial);
		//trial = pow(2, trial);
		trial = (__int64)1 << trial;
		experiment = trial*THREADS*BLOCKS;
		for (int m = 0; m < keys; m++) {
//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(key_d, key, 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				ASCON4 << <BLOCKS, THREADS >> >(key_d, nonce_d, key_choice, counter_d);
			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
		}
	}
	else if (choice == 11) {
		printf("Select key (1-4): ");
		scanf_s("%d", &key_choice);
		printf("Trial = 2^15 +  ");
		scanf_s("%d", &trial);
		//trial = pow(2, trial);
		trial = (__int64)1 << trial;
		experiment = trial*THREADS*BLOCKS;
		printf("Experiment: %I64d\n", experiment);
		for (int m = 0; m < keys; m++) {
//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(key_d, key, 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				ASCON5 << <BLOCKS, THREADS >> >(key_d, nonce_d, key_choice, counter_d);
			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			average_bias += bias;
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
		}
		average_bias /= keys;
		printf("Average bias: %I64d\n",average_bias);
	}
	else if (choice == 12) {
		printf("Select key (1-4): ");
		scanf_s("%d", &key_choice);
		experiment = trial*TRIALS*THREADS*BLOCKS;
		printf("Experiment: %I64d\n", experiment);
		for (int m = 0; m < keys; m++) {
//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(key_d, key, 2 * sizeof(bit64), hipMemcpyHostToDevice);
				ASCON5b << <BLOCKS, THREADS >> >(key_d, nonce_d, key_choice, counter_d);

			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			average_bias += bias;
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
		}
		average_bias /= keys;
		printf("Average bias: %I64d\n", average_bias);
	}
	else if (choice == 13) {
		printf("Select key (1-4): ");
		scanf_s("%d", &key_choice);
		printf("Rotation (0-63): ");
		scanf_s("%d", &rotation);
		experiment = trial*TRIALS*THREADS*BLOCKS;
		printf("Experiment: %I64d\n", experiment);
		for (int m = 0; m < keys; m++) {
//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(key_d, key, 2 * sizeof(bit64), hipMemcpyHostToDevice);
				ASCON5c << <BLOCKS, THREADS >> >(key_d, nonce_d, key_choice, counter_d, rotation);

			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			average_bias += bias;
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
		}
		average_bias /= keys;
		printf("Average bias: %I64d\n", average_bias);
	}
	else if (choice == 14) {
		printf("Select key (1-4): ");
		scanf_s("%d", &key_choice);
		printf("Rotation (0-63): ");
		scanf_s("%d", &rotation);
		experiment = trial*TRIALS*THREADS*BLOCKS;
		printf("Experiment: %I64d\n", experiment);
		for (int m = 0; m < keys; m++) {
//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(key_d, key, 2 * sizeof(bit64), hipMemcpyHostToDevice);
				ASCON6 << <BLOCKS, THREADS >> >(key_d, nonce_d, key_choice, counter_d, rotation);

			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			average_bias += bias;
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
		}
		average_bias /= keys;
		printf("Average bias: %I64d\n", average_bias);
	}
	else if (choice == 15) {
		FILE *fp;
		int shift = 0, flag=0, device_select=0;
		printf("Select key (1-4): ");
		scanf_s("%d", &key_choice);
		printf("Pairs (2 ^ 35 + ?): ");
		scanf_s("%d", &shift);
		printf("Select Cuda Device: ");
		scanf_s("%d", &device_select);
		hipSetDevice(device_select);
		trial = 1;		trial = trial << shift;
		experiment = trial*TRIALS*THREADS*BLOCKS;
		if (key_choice == 1) fopen_s(&fp, "Automatic_search_key1.txt", "w");
		if (key_choice == 2) fopen_s(&fp, "Automatic_search_key2.txt", "w");
		if (key_choice == 3) fopen_s(&fp, "Automatic_search_key3.txt", "w");
		if (key_choice == 4) fopen_s(&fp, "Automatic_search_key4.txt", "w");
		printf("Key Choice: %d\n", key_choice); fprintf(fp,"Key Choice: %d\n", key_choice);
		printf("Pairs: 2 ^ %d\n", shift + 35); fprintf(fp,"Pairs: 2 ^ %d\n", shift + 35);
		printf("Experiment: %I64d\n", experiment); fprintf(fp,"Experiment: %I64d\n", experiment);
		for (int rotation = 0; rotation < 64; rotation++){
			total_counter = 0; bias = 0; average_bias = 0; flag = 0;
			printf("Rotation: %d\n", rotation); fprintf(fp, "Rotation: %d\n", rotation);
			for (int m = 0; m < keys; m++) {
//				__int64 counter[BLOCKS * THREADS] = { 0 };
				__int64 *counter;
				counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
				total_counter = 0;
				hipMalloc((void **)&key_d, 2 * sizeof(bit64));
				hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
				hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
				rdrand_64(key, 0);
				rdrand_64(key + 1, 0);
				//		printf("%I64x %I64x\n", key[0], key[1]);
				StartCounter();
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				for (int i = 0; i < trial; i++) {
					for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); }
					hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
					hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
					hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
					hipMemcpy(key_d, key, 2 * sizeof(bit64), hipMemcpyHostToDevice);
					ASCON6 << <BLOCKS, THREADS >> >(key_d, nonce_d, key_choice, counter_d, rotation);

				}
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				printf("Time of kernel: %lf\n", GetCounter());
				for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
				bias = (experiment) / 2 - total_counter;
				printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
				fprintf(fp,"%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
				average_bias += bias;
				hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
				if (bias > 0 && flag < 0) m = keys + 1;
				else if (bias < 0 && flag > 0) m = keys + 1;
				if (bias > 0) flag = 1;
				else if (bias < 0) flag = -11;
			}
			average_bias /= keys;
			printf("Average bias: %I64d\n", average_bias); fprintf(fp, "Average bias: %I64d\n", average_bias);
		}
		fclose(fp);
	}
	else if (choice == 20) {
		FILE* fp;
		fopen_s(&fp, "eprint6r.txt", "w");
		printf("Trial = 2^35 +  ");
		scanf_s("%d", &trial);
		//trial = pow(2, trial);
		trial = (__int64)1 << trial;
		experiment = TRIALS*trial*THREADS*BLOCKS;
		for (int m = 0; m < 1; m++) {
			//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			StartCounter();
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
				for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				ASCON6_eprint << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, counter_d);
			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			printf("Time of kernel: %lf\n", GetCounter());
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			fprintf(fp,"%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
			fclose(fp);
		}
	}
	else if (choice == 100) {
		FILE* fp;
		fopen_s(&fp, "eprint1r.txt", "ab");
		printf("Trial = 2^35 +  ");
		scanf_s("%d", &trial);
		//trial = pow(2, trial);
		trial = (__int64)1 << trial;
		experiment = TRIALS*trial*THREADS*BLOCKS;
		printf("Trial: %I64d Experiment: %I64d\n", trial, experiment);
		fprintf(fp, "Trial: %I64d Experiment: %I64d\n", trial, experiment);
		for (int m = 0; m < 1; m++) {
			//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			StartCounter();
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
				for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				ASCON1_eprint << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, counter_d);
			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			printf("Time of kernel: %lf\n", GetCounter());
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			fprintf(fp, "%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
			fclose(fp);
		}
	}
	else if (choice == 40) {
		printf("Trial = 2^35 +  ");
		scanf_s("%d", &trial);
		//trial = pow(2, trial);
		trial = (__int64)1 << trial;
		experiment = TRIALS*trial*THREADS*BLOCKS;
		for (int m = 0; m < keys; m++) {
//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			StartCounter();
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
				for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				ASCON4_eprint << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, counter_d);
			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			printf("Time of kernel: %lf\n", GetCounter());
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
		}
	}
	else if (choice == 50) {
		FILE* fp;
		fopen_s(&fp, "hossein2r.txt", "w");
		printf("Trial = 2^35 +  ");
		scanf_s("%d", &trial);
		//trial = pow(2, trial);
		trial = (__int64)1 << trial;
		experiment = TRIALS*trial*THREADS*BLOCKS;
		for (int m = 0; m < 1; m++) {
			//			__int64 counter[BLOCKS * THREADS] = { 0 };
			__int64 *counter;
			counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
			total_counter = 0;
			hipMalloc((void **)&key_d, 2 * sizeof(bit64));
			hipMalloc((void **)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&keyrows_d, BLOCKS * THREADS * 2 * sizeof(bit64));
			hipMalloc((void **)&IV_d, BLOCKS * THREADS * sizeof(bit64));
			hipMalloc((void **)&counter_d, BLOCKS * THREADS * sizeof(bit64));
			rdrand_64(key, 0);
			rdrand_64(key + 1, 0);
			//		printf("%I64x %I64x\n", key[0], key[1]);
			StartCounter();
			hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
			for (int i = 0; i < trial; i++) {
				for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); rdrand_64(keyrows + j, 0); }
				for (int j = 0; j < THREADS * BLOCKS; j++) { rdrand_64(IV + j, 0); }
				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(keyrows_d, keyrows, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(IV_d, IV, BLOCKS * THREADS * sizeof(bit64), hipMemcpyHostToDevice);
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
				ASCON2_hossein << <BLOCKS, THREADS >> >(IV_d, keyrows_d, nonce_d, counter_d);
			}
			hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
			printf("Time of kernel: %lf\n", GetCounter());
			for (int i = 0; i < BLOCKS*THREADS; i++) total_counter += counter[i];
			bias = (experiment) / 2 - total_counter;
			printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			fprintf(fp, "%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
			hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
			fclose(fp);
		}
	}
	else if (choice == 200) ASCON_benchmark();
	else if (choice == 201)	ASCON55_crypto24();
	free(nonce); free(keyrows); free(IV);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
}

